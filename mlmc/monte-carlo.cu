#include "hip/hip_runtime.h"


////////////////////////////////////////////////////////////////////////
// GPU version of Monte Carlo algorithm using NVIDIA's CURAND library
////////////////////////////////////////////////////////////////////////

#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hip/hip_fp16.h>

#include "fp16_conversion"
#include "hip/hip_runtime_api.h"

////////////////////////////////////////////////////////////////////////
// CUDA global constants
////////////////////////////////////////////////////////////////////////

__constant__ int   N;
__constant__ float T, r, sigma, rho, alpha, dt, con1, con2;

////////////////////////////////////////////////////////////////////////
// kernel routine basic. As provided in practical 2.
////////////////////////////////////////////////////////////////////////


__global__ void pathcalc(float *d_z, float *d_v)
{
  float s1, s2, y1, y2, payoff;
  int   ind;

  // move array pointers to correct position

  // version 1
  ind = threadIdx.x + 2*N*blockIdx.x*blockDim.x;


  // path calculation

  s1 = 1.0f;
  s2 = 1.0f;

  for (int n=0; n<N; n++) {
    y1   = d_z[ind];
    ind += blockDim.x;      // shift pointer to next element

    y2   = rho*y1 + alpha*d_z[ind];
    ind += blockDim.x;      // shift pointer to next element

    s1 = s1*(con1 + con2*y1);
    s2 = s2*(con1 + con2*y2);
  }

  // put payoff value into device array
  payoff = 0.0f;
  if ( fabs(s1-1.0f)<0.1f && fabs(s2-1.0f)<0.1f ) payoff = exp(-r*T);

  d_v[threadIdx.x + blockIdx.x*blockDim.x] = payoff;
}

////////////////////////////////////////////////////////////////////////
// kernel routine level path calc. 
////////////////////////////////////////////////////////////////////////

#define pathcalc_low pathcalc_level<1>
#define pathcalc_mid pathcalc_level<2>
#define pathcalc_high pathcalc_level<3>

template <int level>
__global__ void pathcalc_level(float *d_z, float *d_v)
{
  float s1, s2, y1, y2, payoff;
  int   ind;

  // move array pointers to correct position

  // version 1
  ind = threadIdx.x + 2*N*blockIdx.x*blockDim.x;


  // path calculation

  s1 = 1.0f;
  s2 = 1.0f;

  for (int n=0; n<N; n++) {
    y1   = d_z[ind];
    ind += blockDim.x;      // shift pointer to next element

    y2   = rho*y1 + alpha*d_z[ind];
    ind += blockDim.x;      // shift pointer to next element

    s1 = s1*(con1 + con2*y1);
    s2 = s2*(con1 + con2*y2);
  }

  // put payoff value into device array
  payoff = 0.0f;
  if ( fabs(s1-1.0f)<0.1f && fabs(s2-1.0f)<0.1f ) payoff = exp(-r*T);

  d_v[threadIdx.x + blockIdx.x*blockDim.x] = payoff;
}

////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////

/*
   P = mlmc(Lmin,Lmax,N0,eps, mlmc_l, alpha,beta,gamma, Nl)
 
   multilevel Monte Carlo control routine

   num_levels  = levels of refinement       >= 2
   n_initial    = initial number of samples         > 0
   epsilon   = desired accuracy (rms error)      > 0 
 
   alpha -> weak error is  O(2^{-alpha*l})
   beta  -> variance is    O(2^{-beta*l})
   gamma -> sample cost is O(2^{gamma*l})

   if alpha, beta, gamma are not positive then they will be estimated

   mlmc_l(l,N,sums)   low-level function
        l       = level
        N       = number of paths
        sums[0] = sum(cost)
        sums[1] = sum(Y)
        sums[2] = sum(Y.^2)
        where Y are iid samples with expected value:
        E[P_0]           on level 0
        E[P_l - P_{l-1}] on level l>0

   P     = value
   Nl    = number of samples at each level
   NlCl  = total cost of samples at each level

*/

int mlmc_gpu(
	int num_levels,
	int n_initial, float epsilon, 
	float alpha_0, float beta_0, float gamma_0, 
	int &out_samples_per_level, float &out_cost_per_level,
	bool use_debug, bool use_timings)
			{
    
  int     NPATH=960000, h_N=100;
  float   h_T, h_r, h_sigma, h_rho, h_alpha, h_dt, h_con1, h_con2;
  float  *h_v, *d_v, *d_z;
  double  sum1, sum2;

  // initialise card

  const char * arg = "hi";
  const char ** argv = &arg;
  findCudaDevice(0, argv);

  // initialise CUDA timing

  float milli;
  hipEvent_t start, stop;

  if (use_timings) {
    hipEventCreate(&start);
    hipEventCreate(&stop);
  }

  // allocate memory on host and device

  h_v = (float *)malloc(sizeof(float)*NPATH);

  checkCudaErrors( hipMalloc((void **)&d_v, sizeof(float)*NPATH) );
  checkCudaErrors( hipMalloc((void **)&d_z, sizeof(float)*2*h_N*NPATH) );

  // define constants and transfer to GPU

  h_T     = 1.0f;
  h_r     = 0.05f;
  h_sigma = 0.1f;
  h_rho   = 0.5f;
  h_alpha = sqrt(1.0f-h_rho*h_rho);
  h_dt    = 1.0f/h_N;
  h_con1  = 1.0f + h_r*h_dt;
  h_con2  = sqrt(h_dt)*h_sigma;

  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(N),    &h_N,    sizeof(h_N)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(T),    &h_T,    sizeof(h_T)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(r),    &h_r,    sizeof(h_r)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(sigma),&h_sigma,sizeof(h_sigma)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(rho),  &h_rho,  sizeof(h_rho)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(alpha),&h_alpha,sizeof(h_alpha)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(dt),   &h_dt,   sizeof(h_dt)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(con1), &h_con1, sizeof(h_con1)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(con2), &h_con2, sizeof(h_con2)) );

  // random number generation

  // - all random numbers are generated before hand and stored in
  // the array d_z, which will be used by the kernel.
  // Kernel speed then depends on the method of which indexing method
  // to access by.

  if (use_timings)
    hipEventRecord(start);

  hiprandGenerator_t gen;
  checkCudaErrors( hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT) );
  checkCudaErrors( hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL) );
  checkCudaErrors( hiprandGenerateNormal(gen, d_z, 2*h_N*NPATH, 0.0f, 1.0f) );
 
  if (use_timings) {
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milli, start, stop);

    printf("CURAND normal RNG  execution time (ms): %f,  samples/sec: %e \n",
            milli, 2.0*h_N*NPATH/(0.001*milli));
  }

  // execute kernel and time it

  hipEventRecord(start);

  pathcalc_low<<<NPATH/64, 64>>>(d_z, d_v);
  getLastCudaError("pathcalc execution failed\n");

  if (use_timings) {
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milli, start, stop);

    printf("Monte Carlo kernel execution time (ms): %f \n",milli);
  }

  // copy back results
  
  checkCudaErrors( hipMemcpy(h_v, d_v, sizeof(float)*NPATH,
                   hipMemcpyDeviceToHost) );

  // compute average

  sum1 = 0.0;
  sum2 = 0.0;
  for (int i=0; i<NPATH; i++) {
    sum1 += h_v[i];
    sum2 += h_v[i]*h_v[i];
  }

  printf("\nAverage value and standard deviation of error  = %13.8f %13.8f\n\n",
	 sum1/NPATH, sqrt((sum2/NPATH - (sum1/NPATH)*(sum1/NPATH))/NPATH) );

  // Tidy up library

  checkCudaErrors( hiprandDestroyGenerator(gen) );

  // Release memory and exit cleanly

  free(h_v);
  checkCudaErrors( hipFree(d_v) );
  checkCudaErrors( hipFree(d_z) );

  // CUDA exit -- needed to flush printf write buffer

  hipDeviceReset();

  return 0;
}
