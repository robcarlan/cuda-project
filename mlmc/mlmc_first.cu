#include "hip/hip_runtime.h"

////////////////////////////////////////////////////////////////////////
// GPU version of Monte Carlo algorithm using NVIDIA's CURAND library,
// With annotations.
////////////////////////////////////////////////////////////////////////

#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hiprand.h>

//#if __CUDA_ARCH__ >= 530
#include <hip/hip_fp16.h>
//#pragma message("Compiling with CUDA half precision")
//#else
//#warning "CUDA half precision not supported!"
//#endif

#include <hip/hip_runtime_api.h>

void regression(int, float *, float *, float &a, float &b);
void Check_CUDA_Error(const char *message);
int round_to_grid_size(int grid_size, int n);

float mlmc_gpu(
	int num_levels,
	int n_initial, float eps,
	float alpha_0, float beta_0, float gamma_0,
	int *out_samples_per_level, float *out_cost_per_level,
	int debug_level, bool use_timings, 
	bool gpu_reduce, bool milstein);
	
int round_to_grid_size(int grid_size, int n) {
	if (n % grid_size == 0) return n;
	else return (n / grid_size + 1)  * grid_size;
}

void Check_CUDA_Error(const char *message) {
	hipError_t error = hipGetLastError();
	if(error!=hipSuccess) {
		fprintf(stderr,"ERROR: %s: %s\n", message, 
		hipGetErrorString(error) );
		exit(-1);
	}
}

////////////////////////////////////////////////////////////////////////
// CUDA global constants
////////////////////////////////////////////////////////////////////////

__constant__ int   N;
// Store double constants and replace with kernel casts
__constant__ double T_dbl, r_dbl, sigma_dbl, rho_dbl, alpha_dbl, dt_dbl, con1_dbl, con2_dbl;

////////////////////////////////////////////////////////////////////////
// kernel routine
////////////////////////////////////////////////////////////////////////

// Attempts to solve 2D geometric brownian motion SDE:
//   dS_1 = r * S_1 * dt + rho * S_1 * dW_1
//   dS_2 = r * S_2 * dt + rho * S_2 * dW_2
// dW_1 and dW_2 are increments in two correlated brownian motions.

// This is approximated using Euler-Maruyama discretisation:
//   S_1,N+1 = S_1,N * (1 + r*delta(T) + rho * sqrt(delta(T)) * Y_1, N
//   S_2,N+1 = S_2,N * (1 + r*delta(T) + rho * sqrt(delta(T)) * Y_2, N

// delta(T) is the timestep. Y1_N and Y2_N are Normal r.v.
// Independent with other timesteps, but have correlation p which can be simulated by defining them as:
//   Y_1,N = Z_1,N
//   Y_2,N = rho * Z_1,N + sqrt(1-rho^2) * Z_2,N

// Note that the more complex version mlqmc06_l uses a Milstein method of discretisation which adds another term to Euler disc.
// Euler has strong order of convergence sqrt(delta(t)) compared to Milsteins delta(T)

__global__ void sum_reduce(double *d_v, double *d_v_sq) {
	extern __shared__  float temp[];
	extern __shared__  float tempsq[];
	
    int tid = threadIdx.x + blockIdx.x*blockDim.x;

    // first, each thread loads data into shared memory
    temp[tid] = d_v[tid];
	tempsq[tid] = d_v_sq[tid];
	
	__syncthreads();

    // next, we perform binary tree reduction

    for (int d = blockDim.x>>1; d > 0; d >>= 1) {
      __syncthreads();  // ensure previous step completed 
      if (tid<d)  {
		  temp[tid] += temp[tid+d];
		  tempsq[tid] += tempsq[tid+d];
	  }
    }

    // finally, first thread puts result into global memory

    if (tid==0){
		d_v[0] = temp[0];
		d_v_sq[0] = tempsq[0];
	} 
}

//#if __CUDA_ARCH__ >= 530
__global__ void pathcalc_half(float *d_z, double *d_v, double *d_v_sq)
{

  __half one = __float2half(1.0f);
  __half point1 = __float2half(0.1f);
  __half negpoint1 = __float2half(-0.1f);

  __half s1, s2, y1, y2;
  float payoff = 0.0f;
  int   ind;

  // move array pointers to correct position

  ind = threadIdx.x + 2*N*blockIdx.x*blockDim.x;

  // path calculation

  s1 = one;
  s2 = s1;

  for (int n=0; n<N; n++) {
    y1   = __float2half(d_z[ind]);
    ind += blockDim.x;      // shift pointer to next element

    y2   = __hfma(__float2half((float)rho_dbl), y1,
		  __hmul(__float2half((float)alpha_dbl), __float2half(d_z[ind])));
    ind += blockDim.x;      // shift pointer to next element

    s1 = __hmul(s1, (__hfma(__float2half((float)con2_dbl), y1, __float2half((float)con1_dbl))));
    s2 = __hmul(s2, (__hfma(__float2half((float)con2_dbl), y2, __float2half((float)con1_dbl))));
  
  }

  // put payoff value into device array

  payoff = 0.0f;
  __half s1diff = __hsub(s1, one);
  __half s2diff = __hsub(s2, one);

  if ( 	__hgt(s1diff, negpoint1) && __hlt(s1diff, point1) &&
		__hgt(s2diff, negpoint1) && __hlt(s2diff, point1) )
      payoff = __half2float(hexp(__hmul(__float2half((float)-r_dbl),
					__float2half((float)T_dbl))) );
  
  d_v[threadIdx.x + blockIdx.x*blockDim.x] = payoff;
  d_v_sq[threadIdx.x + blockIdx.x*blockDim.x] = payoff * payoff;
}
//#else 
//#define pathcalc_half pathcalc_float
//#endif

//#if __CUDA_ARCH__ >= 530
__global__ void pathcalc_float(float *d_z, double *d_v, double *d_v_sq)
{
  float s1, s2, y1, y2, payoff, payoffh;
  __half s1h, s2h, y1h, y2h;
  int   ind;

  // move array pointers to correct position
  ind = threadIdx.x + 2*N*blockIdx.x*blockDim.x;
  
  __half one = __float2half(1.0f);
  __half point1 = __float2half(0.1f);
  __half negpoint1 = __float2half(-0.1f);

  // path calculation
  s1 = 1.0f;
  s2 = 1.0f;
  s1h = one;
  s2h = s1h;
  
  for (int n=0; n<N; n++) {
    y1   = d_z[ind];
    ind += blockDim.x;      // shift pointer to next element

    y2   = rho_dbl *y1 + alpha_dbl * d_z[ind];
    ind += blockDim.x;      // shift pointer to next element

    s1 = s1*(con1_dbl + con2_dbl *y1);
    s2 = s2*(con1_dbl + con2_dbl *y2);
	
    y1h   = __float2half(d_z[ind]);
    y2h   = __hfma(__float2half((float)rho_dbl), y1h,
		  __hmul(__float2half((float)alpha_dbl), __float2half(d_z[ind])));
    s1h = __hmul(s1h, (__hfma(__float2half((float)con2_dbl), y1h, __float2half((float)con1_dbl))));
    s2h = __hmul(s2h, (__hfma(__float2half((float)con2_dbl), y2h, __float2half((float)con1_dbl))));
	
  }

  // put payoff value into device array

  payoff = 0.0f;
  payoffh = 0.0f;
  if ( fabs(s1-1.0f)<0.1f && fabs(s2-1.0f)<0.1f ) payoff = exp(-r_dbl * T_dbl);
  
  __half s1diff = __hsub(s1h, one);
  __half s2diff = __hsub(s2h, one);
  
  if (__hgt(s1diff, negpoint1) && __hlt(s1diff, point1) &&
	  __hgt(s2diff, negpoint1) && __hlt(s2diff, point1) )
	payoffh = __half2float(
					hexp(__hmul(
							__float2half((float)(-r_dbl)),
							__float2half((float)T_dbl))));		


  d_v[threadIdx.x + blockIdx.x*blockDim.x] = payoff - payoffh;
  d_v_sq[threadIdx.x + blockIdx.x*blockDim.x] = (payoff - payoffh) * (payoff - payoffh);

}
//#else 
//__global__ void pathcalc_float(float *d_z, double *d_v, double *d_v_sq)
//{
	//As estimators would be the same! Not great at all.
//  d_v[threadIdx.x + blockIdx.x*blockDim.x] = 0.0;
//  d_v_sq[threadIdx.x + blockIdx.x*blockDim.x] = 0.0;
//}	
//#endif

template <bool gpu_reduce>
__global__ void pathcalc_double(float *d_z, double *d_v, double *d_v_sq)
{
  double s1, s2, y1, y2, payoff;
  float s1f, s2f, y1f, y2f, payofff;
  int   ind;

  // move array pointers to correct position
  ind = threadIdx.x + 2*N*blockIdx.x*blockDim.x;

  // path calculation
  s1 = 1.0;
  s2 = 1.0;
  s1f = 1.0;
  s2f = 1.0;

  for (int n=0; n<N; n++) {
    y1   = d_z[ind];
	y1f   = d_z[ind];
    ind += blockDim.x;      // shift pointer to next element

    y2   = rho_dbl *y1 + alpha_dbl *d_z[ind];
	y2f   = (float)rho_dbl * y1f + alpha_dbl * d_z[ind];
    ind += blockDim.x;      // shift pointer to next element

    s1 = s1*(con1_dbl + con2_dbl *y1);
    s2 = s2*(con1_dbl + con2_dbl *y2);
	s1f = s1f*(con1_dbl + con2_dbl *y1f);
    s2f = s2f*(con1_dbl + con2_dbl *y2f);
  }

  // put payoff value into device array

  payoff = payofff = 0.0f;
  if ( abs(s1-1.0)<0.1 && abs(s2-1.0)<0.1 ) payoff = exp(-r_dbl * T_dbl);
  if ( abs(s1f-1.0)<0.1 && abs(s2f-1.0)<0.1 ) payofff = exp(-r_dbl * T_dbl);

  d_v[threadIdx.x + blockIdx.x*blockDim.x] = payoff - payofff;
  d_v_sq[threadIdx.x + blockIdx.x*blockDim.x] = (payoff - payofff) * (payoff - payofff);  
  
  //printf("Dif: %g \n",d_v[threadIdx.x + blockIdx.x*blockDim.x] );
}

void pathcalc(int level, int gsize, int samples, float *d_z, double *d_v, double *d_v_sq,
		bool gpu_reduce, bool use_milstein) {
	if (!use_milstein){
		if (level == 0)
			pathcalc_half<<<samples / gsize, gsize>>>(d_z, d_v, d_v_sq);
		if (level == 1)
			pathcalc_float<<<samples / gsize, gsize>>>(d_z, d_v, d_v_sq);
		if (level == 2)
			pathcalc_double<false><<<samples / gsize, gsize>>>(d_z, d_v, d_v_sq);
		 Check_CUDA_Error("Euler Kernel Execution Failed!\n");
		 
	} else {
		pathcalc_float<<<samples / gsize, gsize>>>(d_z, d_v, d_v_sq);
		
		 Check_CUDA_Error("Milstein Kernel Execution Failed!\n");
	}
	
	if (gpu_reduce) {
		sum_reduce<<<samples / gsize, gsize, 2 * sizeof(double)>>>(d_v, d_v_sq);
		
		 Check_CUDA_Error("GPU Reduce Kernel Failed!\n");
		 
		 //Sum over blocks.
		 
		 //Optimisation - threadfence shuffles and stuff kernelise blocks
	}
}

////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////

float mlmc_gpu(
	int num_levels,
	int n_initial, float eps,
	float alpha_0, float beta_0, float gamma_0,
	int *out_samples_per_level, float *out_cost_per_level,
	int debug_level, bool use_timings, 
	bool gpu_reduce, bool milstein)
{
    int *Nl = out_samples_per_level;
    float *Cl = out_cost_per_level;
	
	if (gpu_reduce) printf("GPU reduce");

  if (debug_level) {
      printf("CUDA multi-level monte carlo variant 1\n");
#if __CUDA_ARCH__ >= 520
      printf("(CUDA half precision enabled...)\n");
#else
      printf("(CUDA half precision NOT enabled...)\n");
#endif
  }

  //This variant sets LMin and LMax set to be 2.
  int Lmin = 2;
  int Lmax = 2;
  int grid_size = 64;

  //Number of timesteps.
  int h_N = 100;
  
  n_initial = round_to_grid_size(grid_size, n_initial);

  double   h_T, h_r, h_sigma, h_rho, h_alpha, h_dt, h_con1, h_con2;

  h_T     = 1.0;
  h_r     = 0.05;
  h_sigma = 0.1;
  h_rho   = 0.5;
  h_alpha = sqrt(1.0-h_rho*h_rho);
  h_dt    = 1.0/ 128;
  h_con1  = 1.0 + h_r*h_dt;
  h_con2  = sqrt(h_dt)*h_sigma;

  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(N),                &h_N,                           sizeof(h_N)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(T_dbl),    	&h_T,    			sizeof(h_T)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(r_dbl),    	&h_r,    			sizeof(h_r)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(sigma_dbl),	&h_sigma,			sizeof(h_sigma)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(rho_dbl),  	&h_rho,  			sizeof(h_rho)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(alpha_dbl ),	&h_alpha,			sizeof(h_alpha)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(dt_dbl),   	&h_dt,   			sizeof(h_dt)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(con1_dbl), 	&h_con1, 			sizeof(h_con1)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(con2_dbl), 	&h_con2, 			sizeof(h_con2)) );

  double sums[7], suml[3][21];
  float  ml[21], Vl[21], NlCl[21], x[21], y[21],
         alpha, beta, gamma, sum, theta;
  int    dNl[21], L, converged;

  int    diag = debug_level;  // diagnostics, set to 0 for none
  
  //
  // check input parameters
  //

  if (num_levels < 1) {
    fprintf(stderr,"error: needs num_levels >= 1 \n");
    exit(1);
  }

  //
  // initialisation
  //

  // initialise CUDA timing

  float milli;
  hipEvent_t start, stop;

  if (use_timings) {
    hipEventCreate(&start);
    hipEventCreate(&stop);
  }

  hiprandGenerator_t gen;
  checkCudaErrors( hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT) );
  checkCudaErrors( hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL) );
	
  alpha = fmax(0.0f,alpha_0);
  beta  = fmax(0.0f,beta_0);
  gamma = fmax(0.0f,gamma_0);
  theta = 0.25f;             // MSE split between bias^2 and variance

  L = Lmin;
  converged = 0;

  for(int l=0; l<=Lmax; l++) {
    Nl[l]   = 0;
    Cl[l]   = powf(2.0f,(float)l*gamma);
    NlCl[l] = 0.0f;

    for(int n=0; n<3; n++) suml[n][l] = 0.0;
  }

  for(int l=0; l<=Lmin; l++) dNl[l] = n_initial;

  if (diag > 1)
      printf("Initialised - entering main loop.\n");

  //
  // main loop
  //

  while (!converged) {

    //
    // update sample sums
    //

    for (int l=0; l<=L; l++) {
      if (diag) printf(" %d ",dNl[l]);

      if (dNl[l]>0) {
		  
		sums[0] = sums[1] = sums[2] = 0.0f;

    	int num_paths = dNl[l];

    	double *h_v, *d_v;
    	double *h_v_sq, *d_v_sq;
    	float  *d_z;

    	//Allocate memory
		
		//TODO :: should be the same for each level.
    	h_v = (double *)malloc(sizeof(double) * num_paths);
    	h_v_sq = (double *)malloc(sizeof(double) * num_paths);
    	checkCudaErrors( hipMalloc((void **)&d_v, sizeof(double)*num_paths) );
    	checkCudaErrors( hipMalloc((void **)&d_v_sq, sizeof(double)*num_paths) );
    	checkCudaErrors( hipMalloc((void **)&d_z, sizeof(float)*2*h_N*num_paths) );

		if (debug_level)
			printf("memory initialised level %d\n", l);

		//Generate 2 * dNl[l] random samples at desired precision based on l.
		if (use_timings)
			hipEventRecord(start);

		checkCudaErrors( hiprandGenerateNormal(gen, d_z, 2*h_N*num_paths, 0.0f, 1.0f) );
	 
		if (use_timings) {
			hipEventRecord(stop);
			hipEventSynchronize(stop);
			hipEventElapsedTime(&milli, start, stop);

			printf("CURAND normal RNG  execution time (ms): %f,  samples/sec: %e \n",
			   milli, 2.0*h_N*num_paths/(0.001*milli));
		}

		if (debug_level)
			printf("Runing kernel level %d grid_size %d num_paths %d\n", l, grid_size, num_paths);

		hipDeviceSynchronize();
		pathcalc(l, grid_size, num_paths, d_z, d_v, d_v_sq,
			gpu_reduce, milstein);
		hipDeviceSynchronize();
		
		if (debug_level)
			printf("path calc level %d\n", l);

		//Move results out of device memory, add to array.

		checkCudaErrors( hipMemcpy(h_v, d_v, sizeof(double)*num_paths,
						 hipMemcpyDeviceToHost) );

		checkCudaErrors( hipMemcpy(h_v_sq, d_v_sq, sizeof(double)*num_paths,
						 hipMemcpyDeviceToHost) );

		if (debug_level)
			printf("reduce step\n");

		//reduce step
		if (!gpu_reduce) {
			for (int i = 0; i < num_paths; i++) {
				// Number of timestep is 2^bit precision
				sums[0] += 1 << l;
			
				if (diag > 3)
					printf("[%d,%d] val: %f val_sq: %f\n", l, i, h_v[i], h_v_sq[i]);
			
				sums[1] += h_v[i];
				sums[2] += h_v_sq[i];
			}
		} else {
			//Reduction has already been done on gpu, so just copy over.
			sums[0] += (1 << l) * num_paths;
			sums[1] += h_v[0];
			sums[2] += h_v_sq[0];
		}

		if (debug_level)
			printf("reduce completed %d - %g %g \n", l, sums[1], sums[2]);

		suml[0][l] += (double) num_paths;
		suml[1][l] += sums[1];
		suml[2][l] += sums[2];
		NlCl[l]    += sums[0];  // sum total cost

		//Free used memory
		free(h_v);
		free(h_v_sq);
		checkCudaErrors( hipFree(d_v) );
		checkCudaErrors( hipFree(d_v_sq) );
		checkCudaErrors( hipFree(d_z) );
      }
    }
    if (diag) printf(" \n");

    //
    // compute absolute average, variance and cost,
    // correct for possible under-sampling,
    // and set optimal number of new samples
    //

    sum = 0.0f;

    for (int l=0; l<=L; l++) {
	  printf("ML %g \n" , suml[1][l]);
      ml[l] = fabs(suml[1][l]/suml[0][l]);
      Vl[l] = fmaxf(suml[2][l]/suml[0][l] - ml[l]*ml[l], 0.0f);

      if (diag > 2)
		printf("level %d: variance %.5f expectation %.5f\n", l, Vl[l], ml[l]);
      
      if (gamma_0 <= 0.0f) Cl[l] = NlCl[l] / suml[0][l];

      if (l>1) {
        ml[l] = fmaxf(ml[l],  0.5f*ml[l-1]/powf(2.0f,alpha));
        Vl[l] = fmaxf(Vl[l],  0.5f*Vl[l-1]/powf(2.0f,beta));
      }

      sum += sqrtf(Vl[l]*Cl[l]);
    }

    if (diag > 1) {
		printf("Next level samples: ");
    }
    
    //Now update the number of samples for each level.
    for (int l=0; l<=L; l++) {
      dNl[l] = ceilf( fmaxf( 0.0f,
                       sqrtf(Vl[l]/Cl[l])*sum/((1.0f-theta)*eps*eps)
                     - suml[0][l] ) );
      if (diag > 1) {
		printf(" level %d - %d (rounded to %d), ", l, dNl[l], round_to_grid_size(grid_size, dNl[l]));
      }
	  
	  dNl[l] = round_to_grid_size(grid_size, dNl[l]);
    }
   
    if (diag > 1) {
	printf("\n");
    }
 

    //
    // use linear regression to estimate alpha, beta, gamma if not given
    //

    if (alpha_0 <= 0.0f) {
      for (int l=1; l<=L; l++) {
        x[l-1] = l;
        y[l-1] = - log2f(ml[l]);
      }
      regression(L,x,y,alpha,sum);
      if (diag) printf(" alpha = %f \n",alpha);
    }

    if (beta_0 <= 0.0f) {
      for (int l=1; l<=L; l++) {
        x[l-1] = l;
        y[l-1] = - log2f(Vl[l]);
      }
      regression(L,x,y,beta,sum);
      if (diag) printf(" beta = %f \n",beta);
    }

     if (gamma_0 <= 0.0f) {
      for (int l=1; l<=L; l++) {
        x[l-1] = l;
        y[l-1] = log2f(Cl[l]);
      }
      regression(L,x,y,gamma,sum);
      if (diag) printf(" gamma = %f \n",gamma);
    }

    //
    // if (almost) converged, estimate remaining error and decide
    // whether a new level is required
    //

    sum = 0.0;
      for (int l=0; l<=L; l++)
        sum += fmaxf(0.0f, (float)dNl[l]-0.01f*suml[0][l]);

    if (sum==0) {
      if (diag) printf(" achieved variance target \n");

      converged = 1;
      float rem = ml[L] / (powf(2.0f,gamma)-1.0f);

      if (rem > sqrtf(theta)*eps) {
        if (L==Lmax)
          printf("*** failed to achieve weak convergence *** \n");
        else {
          converged = 0;
          L++;
          Vl[L] = Vl[L-1]/powf(2.0f,beta);
          Cl[L] = Cl[L-1]*powf(2.0f,gamma);

          if (diag) printf(" L = %d \n",L);

          sum = 0.0f;
          for (int l=0; l<=L; l++) sum += sqrtf(Vl[l]*Cl[l]);
          for (int l=0; l<=L; l++) {
            dNl[l] = ceilf( fmaxf( 0.0f,
                            sqrtf(Vl[l]/Cl[l])*sum/((1.0f-theta)*eps*eps)
                          - suml[0][l] ) );
			dNl[l] = round_to_grid_size(grid_size, dNl[l]);
		  }
        }
      }
    }
  }

  //
  // finally, evaluate multilevel estimator and set outputs
  //

  float P = 0.0f;
  for (int l=0; l<=L; l++) {
    P    += suml[1][l]/suml[0][l];
    Nl[l] = suml[0][l];
    Cl[l] = NlCl[l] / Cl[l];
  }

  return P;
}



//
// linear regression routine
//

void regression(int N, float *x, float *y, float &a, float &b){

  float sum0=0.0f, sum1=0.0f, sum2=0.0f, sumy0=0.0f, sumy1=0.0f;

  for (int i=0; i<N; i++) {
    sum0  += 1.0f;
    sum1  += x[i];
    sum2  += x[i]*x[i];

    sumy0 += y[i];
    sumy1 += y[i]*x[i];
  }

  a = (sum0*sumy1 - sum1*sumy0) / (sum0*sum2 - sum1*sum1);
  b = (sum2*sumy0 - sum1*sumy1) / (sum0*sum2 - sum1*sum1);
}

/*

int main(int argc, const char **argv){
    
  int     NPATH=960000, h_N=100;
  float   h_T, h_r, h_sigma, h_rho, h_alpha, h_dt, h_con1, h_con2;
  float  *h_v, *d_v, *d_z;
  double  sum1, sum2;

  // initialise card

  findCudaDevice(argc, argv);

  // initialise CUDA timing

  float milli;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // allocate memory on host and device

  h_v = (float *)malloc(sizeof(float)*NPATH);

  checkCudaErrors( hipMalloc((void **)&d_v, sizeof(float)*NPATH) );
  checkCudaErrors( hipMalloc((void **)&d_z, sizeof(float)*2*h_N*NPATH) );

  // define constants and transfer to GPU

  h_T     = 1.0f;
  h_r     = 0.05f;
  h_sigma = 0.1f;
  h_rho   = 0.5f;
  h_alpha = sqrt(1.0f-h_rho*h_rho);
  h_dt    = 1.0f/h_N;
  h_con1  = 1.0f + h_r*h_dt;
  h_con2  = sqrt(h_dt)*h_sigma;

  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(N),    &h_N,    sizeof(h_N)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(T),    &h_T,    sizeof(h_T)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(r),    &h_r,    sizeof(h_r)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(sigma),&h_sigma,sizeof(h_sigma)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(rho),  &h_rho,  sizeof(h_rho)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(alpha),&h_alpha,sizeof(h_alpha)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(dt),   &h_dt,   sizeof(h_dt)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(con1), &h_con1, sizeof(h_con1)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(con2), &h_con2, sizeof(h_con2)) );

  // random number generation

  hipEventRecord(start);

  hiprandGenerator_t gen;
  checkCudaErrors( hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT) );
  checkCudaErrors( hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL) );
  checkCudaErrors( hiprandGenerateNormal(gen, d_z, 2*h_N*NPATH, 0.0f, 1.0f) );
 
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milli, start, stop);

  printf("CURAND normal RNG  execution time (ms): %f,  samples/sec: %e \n",
          milli, 2.0*h_N*NPATH/(0.001*milli));

  // execute kernel and time it

  hipEventRecord(start);

  //pathcalc<<<NPATH/64, 64>>>(d_z, d_v);
  getLastCudaError("pathcalc execution failed\n");

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milli, start, stop);

  printf("Monte Carlo kernel execution time (ms): %f \n",milli);

  // copy back results

  checkCudaErrors( hipMemcpy(h_v, d_v, sizeof(float)*NPATH,
                   hipMemcpyDeviceToHost) );

  // compute average

  sum1 = 0.0;
  sum2 = 0.0;
  for (int i=0; i<NPATH; i++) {
    sum1 += h_v[i];
    sum2 += h_v[i]*h_v[i];
  }

  printf("\nAverage value and standard deviation of error  = %13.8f %13.8f\n\n",
	 sum1/NPATH, sqrt((sum2/NPATH - (sum1/NPATH)*(sum1/NPATH))/NPATH) );

  // Tidy up library

  checkCudaErrors( hiprandDestroyGenerator(gen) );

  // Release memory and exit cleanly

  free(h_v);
  checkCudaErrors( hipFree(d_v) );
  checkCudaErrors( hipFree(d_z) );

  // CUDA exit -- needed to flush printf write buffer

  hipDeviceReset();

}

*/
